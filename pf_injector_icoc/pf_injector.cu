#include "hip/hip_runtime.h"
#include <sstream>
#include <csignal>
#include <unordered_set>
#include <algorithm>

#include "nvbit_tool.h"
#include "nvbit.h"
#include "utils/utils.h"

#include "globals.h"

#include "pf_injector.h"

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

bool verbose;
__managed__ int verbose_device;
int limit;

// injection parameters input filename: This file is created the script
// that launched error injections
std::string inj_input_filename;
// Output log file
std::string inj_output_filename;

pthread_mutex_t mutex;

__managed__ InjectionInfo inj_info;
__managed__ unsigned long long count_activations_inst[NUM_ISA_INSTRUCTIONS];

std::string last_kernel, last_instruction_sass_str;
uint64_t last_pc_offset;
uint32_t current_instruction_opcode;

template<class ... Ts>
void verbose_printf(Ts &&... inputs) {
    if (verbose) {
        // Do things in your "loop" lambda
        ([&] {
            std::cout << inputs;
        }(), ...);
    }
}

void update_verbose() {
    static bool update_flag = false; // update it only once - performance enhancement
    if (!update_flag) {
        update_flag = true;
        hipDeviceSynchronize();
        verbose_device = verbose;
        hipDeviceSynchronize();
    }
}

void update_inst_counters() {
    static bool update_flag = false;
    if (!update_flag) {
        update_flag = true;
//        CUDA_SAFECALL(hipMallocManaged(&count_activations_inst, sizeof(unsigned long long) * NUM_ISA_INSTRUCTIONS))
        hipDeviceSynchronize();
        std::fill(count_activations_inst, count_activations_inst + NUM_ISA_INSTRUCTIONS, 0);
        hipDeviceSynchronize();
    }
}

uint32_t generate_current_instruction_type(const uint32_t current_opcode) {
    constexpr InstructionType possible_ops[] = {
            FADD, FADD32I, FCMP, FFMA, FFMA32I, FMUL, FMUL32I,
            BFE, BFI, BMSK, BREV, FLO, IABS, IADD, IADD32I,
            IADD3, ICMP, IDP, IDP4A, IMAD, IMAD32I, IMADSP,
            IMUL, IMUL32I, ISAD, ISCADD, ISCADD32I, LOP, LOP32I,
            LOP3, SHF, SHL, SHR, XMAD, ISET, FSET
    };
    std::vector<uint32_t> weights;
    for (auto i: possible_ops) {
        if (i == current_opcode)
            weights.push_back(0);
        else
            weights.push_back(1);
    }

    // gets 'entropy' from device that generates random numbers itself
    // to seed a mersenne twister (pseudo) random generator
    static std::mt19937 generator(std::random_device{}());
    static std::discrete_distribution<> distribution(weights.begin(), weights.end());
    return possible_ops[distribution(generator)];
}

int get_max_regs(hipFunction_t func) {
    int max_regs = -1;
    CUDA_SAFECALL(hipFuncGetAttribute(&max_regs, HIP_FUNC_ATTRIBUTE_NUM_REGS, func))
    return max_regs;
}

void sig_int_handler(int sig) {
    signal(sig, SIG_IGN); // disable Ctrl-C

//    std::ofstream fout(inj_output_filename);
    if (fout.good()) {
        fout << ":::NVBit-inject-error; ERROR FAIL Detected Singal SIGKILL;";
//        fout << " num_activations: " << inj_info.num_activations << ":::";
        fout << inj_info << std::endl;
        fout.flush();
    }
    assert_condition(false, "Ctrl-C pressed, stopping execution!");
}


/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
// DO NOT USE UVM (__managed__) variables in this function
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    // Default values
    verbose = false;
    inj_input_filename = "nvbitfi-injection-info.txt";
    inj_output_filename = "nvbitfi-injection-log-temp.txt";
    limit = INT_MAX;

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default, we instrument everything. */
    auto env_tool_verbose_ptr = std::getenv("TOOL_VERBOSE");
    if (env_tool_verbose_ptr) {
        verbose = std::stoi(std::string(env_tool_verbose_ptr));
    }

    auto env_inj_info_in_file_ptr = std::getenv("INPUT_INJECTION_INFO");
    if (env_inj_info_in_file_ptr) {
        inj_input_filename = env_inj_info_in_file_ptr;
    }

    auto env_inj_info_out_file_ptr = std::getenv("OUTPUT_INJECTION_LOG");
    if (env_inj_info_out_file_ptr) {
        inj_output_filename = env_inj_info_out_file_ptr;
    }

    auto env_instrumentation_limit = std::getenv("INSTRUMENTATION_LIMIT");
    if (env_instrumentation_limit) {
        limit = std::stoi(std::string(env_instrumentation_limit));
    }

    initInstTypeNameMap();
    open_output_file(inj_output_filename);
    assert_condition(fout.good(), "Could not open output file" + inj_output_filename);

    signal(SIGINT, sig_int_handler); // install Ctrl-C handler
    verbose_printf("nvbit_at_init:end\n");
}


void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {

    inj_info.parse_params(inj_input_filename, verbose);  // injParams are updated based on injection seed file
    update_verbose();
    update_inst_counters();

    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);

    // Open the output file
//    std::ofstream fout(inj_output_filename);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f: related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        std::string kname = removeSpaces(nvbit_get_func_name(ctx, f));
        /* Get the vector of instruction composing the loaded CUFunction "func" */
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

        int max_regs = get_max_regs(f);
        assert_condition(fout.good(), "Output file " + inj_output_filename + " not opened");
        fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";max_regs: " << max_regs << "("
             << max_regs << ")" << std::endl;
        for (auto i: instrs) {
            std::string opcode = i->getOpcode();
            std::string inst_type_str = extractInstType(opcode);
            int inst_type = instTypeNameMap[inst_type_str];
            verbose_printf("extracted inst_type: ", inst_type_str, " index of inst_type: ",
                           instTypeNameMap[inst_type_str], "\n");

//            if (inst_type == inj_info.instruction_type_in || inj_info.instruction_type_in == NUM_ISA_INSTRUCTIONS)
            {
                verbose_printf("instruction selected for instrumentation: ");
                if (verbose) {
                    i->print();
                }

                // Tokenize the instruction
                std::vector<std::string> tokens;
                std::string buf; // a buffer string
                std::stringstream ss(i->getSass()); // Insert the string into a stream
                while (ss >> buf)
                    tokens.push_back(buf);

                int dest_GPR_num = -1;
                int num_dest_GPRs = 0;

                if (tokens.size() > 1) { // an actual instruction that writes to either a GPR or PR register
                    verbose_printf("num tokens = ", tokens.size(), "\n");
                    int start = 1; // first token is opcode string
                    if (tokens[0].find('@') != std::string::npos) { // predicated instruction, ignore first token
                        start = 2; // first token is predicate and 2nd token is opcode
                    }

                    // Parse the first operand - this is the first destination
                    int reg_num_1 = -1;
                    int regtype = extractRegNo(tokens[start], reg_num_1);
                    if (regtype == 0) { // GPR reg
                        dest_GPR_num = reg_num_1;
                        auto op_group = getOpGroupNum(inst_type);
                        num_dest_GPRs = (op_group == G_FP64) ? 2 : 1;

                        int sz_str = extractSize(opcode);
                        if (sz_str == 128) {
                            num_dest_GPRs = 4;
                        } else if (sz_str == 64) {
                            num_dest_GPRs = 2;
                        }

                        // Save last instrumented instruction info
                        current_instruction_opcode = inst_type;
                        last_pc_offset = i->getOffset();
                        last_instruction_sass_str = i->getSass();
                        auto is_float = uint32_t(op_group == G_FP32);
                        auto replace_instruction_opcode = generate_current_instruction_type(current_instruction_opcode);
                        auto num_operands = i->getNumOperands();

                        nvbit_insert_call(i, "inject_error", IPOINT_AFTER);
                        nvbit_add_call_arg_const_val64(i, uint64_t(&inj_info));
                        nvbit_add_call_arg_const_val64(i, uint64_t(&verbose_device));
                        nvbit_add_call_arg_const_val64(i, uint64_t(count_activations_inst));
                        // destination GPR register number
                        nvbit_add_call_arg_const_val32(i, dest_GPR_num);
                        // number of destination GPR registers
                        nvbit_add_call_arg_const_val32(i, num_dest_GPRs);
                        // Put if it is float or not
                        nvbit_add_call_arg_const_val32(i, is_float);
                        // Put last opcode index
                        nvbit_add_call_arg_const_val32(i, current_instruction_opcode);
                        // Put the next opcode index
                        nvbit_add_call_arg_const_val32(i, replace_instruction_opcode);
                        //  put the size of the operands at the end of the var list
                        nvbit_add_call_arg_const_val32(i, num_operands);
                        assert_condition(num_operands <= MAX_OPERANDS_NUM,
                                         "More than " + std::to_string(MAX_OPERANDS_NUM) + "operands not managed");

                        /* iterate on the operands */
//                        auto mem_id = 0;
                        for (auto operand_i = num_dest_GPRs; operand_i < num_operands; operand_i++) {
                            /* get the operand_i "i" */
                            const InstrType::operand_t *op = i->getOperand(operand_i);
                            InstrType::OperandType operand_type = op->type;
                            auto casted_operand_type = static_cast<uint32_t>(operand_type);
                            /**
                             * Always put in the following order
                             * 1 operand type const 32 bits
                             * 2 if the operand is valid const 32bits (0 or 1)
                             * 3 operand val, can be 32 bits or mem ref 64 bits
                             */
                            nvbit_add_call_arg_const_val32(i, casted_operand_type, true);
//                            verbose_printf("casted_operand_type ", casted_operand_type, "\nnum_dest_GPRS ",
//                                           num_dest_GPRs, " num operands ", num_operands);
                            switch (operand_type) {
                                case InstrType::OperandType::REG: {
                                    nvbit_add_call_arg_const_val32(i, 1, true);
                                    nvbit_add_call_arg_reg_val(i, op->u.reg.num, true);
                                    break;
                                }
                                case InstrType::OperandType::CBANK: {
                                    nvbit_add_call_arg_const_val32(i, 1, true);
                                    if (op->u.cbank.has_imm_offset) {
                                        nvbit_add_call_arg_cbank_val(i, op->u.cbank.id, op->u.cbank.imm_offset, true);
                                    } else {
                                        nvbit_add_call_arg_cbank_val(i, op->u.cbank.id, op->u.cbank.reg_offset, true);
                                    }
                                    break;
                                }
                                case InstrType::OperandType::MREF:
//                                {
//                                    nvbit_add_call_arg_const_val32(i, 1, true);
////                                    verbose_printf("HAS RA ", op->u.mref.has_ra, " has mmr ", op->u.mref.has_imm, "\n");
//                                    nvbit_add_call_arg_const_val32(i, op->u.mref.has_ra, true);
//                                    nvbit_add_call_arg_const_val32(i, op->u.mref.has_imm, true);
//                                    if (op->u.mref.has_ra){
//                                        nvbit_add_call_arg_reg_val(i, op->u.mref.ra_num, true);
//                                    }
//                                    if (op->u.mref.has_imm){
//                                        assert_condition(mem_id == 0, "Interesting case here\n");
//                                        nvbit_add_call_arg_mref_addr64(i, mem_id, true);
//                                        mem_id++;
//                                    }
//                                    break;
//                                }
                                case InstrType::OperandType::IMM_UINT64:
                                case InstrType::OperandType::IMM_DOUBLE:
                                case InstrType::OperandType::GENERIC:
                                case InstrType::OperandType::UREG:
                                case InstrType::OperandType::UPRED:
                                case InstrType::OperandType::PRED: {
                                    nvbit_add_call_arg_const_val32(i, 0, true);
                                    nvbit_add_call_arg_const_val32(i, 0, true);
                                    break;
                                }
                            }
                        }
                    }
                    // If an instruction has two destination registers, not handled!! (TODO: Fix later)
                }
            }
        }
    }
}

/* This call-back is triggered every time a CUDA event is encountered.
 * Here, we identify CUDA kernel launch events and reset the "counter" before
 * th kernel is launched, and print the counter after the kernel has completed
 * (we make sure it has completed by using hipDeviceSynchronize()). To
 * selectively run either the original or instrumented kernel we used
 * nvbit_enable_instrumented() before launching the kernel. */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch ||
        cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid ||
        cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel) {

//        std::ofstream fout(inj_output_filename);

        /* cast params to cuLaunch_params since if we are here we know these are
         * the right parameters type */
        auto *p = (cuLaunch_params *) params;

        if (!is_exit) {
            pthread_mutex_lock(&mutex);
            if (kernel_id < limit) {
                instrument_function_if_needed(ctx, p->f);
                // hipDeviceSynchronize();

                nvbit_enable_instrumented(ctx, p->f, true); // run the instrumented version
                // hipDeviceSynchronize();
            } else {
                nvbit_enable_instrumented(ctx, p->f, false); // do not use the instrumented version
            }

        } else {
            if (kernel_id < limit) {
                verbose_printf("is_exit\n");
                hipDeviceSynchronize();

                hipError_t le = hipGetLastError();

                std::string kname = removeSpaces(nvbit_get_func_name(ctx, p->f));
                unsigned num_ctas = 0;
                if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
                    cbid == API_CUDA_cuLaunchKernel) {
                    auto *p2 = (cuLaunchKernel_params *) params;
                    num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
                }
                assert_condition(fout.good(), "Output file " + inj_output_filename + " not opened");

                fout << "Injection data; ";
                fout << "index: " << kernel_id << ";";
                fout << "kernel_name: " << kname << ";";
                fout << "ctas: " << num_ctas << ";";
                fout << inj_info << std::endl;
                last_kernel = kname;

                if (hipSuccess != le) {
                    assert_condition(fout.good(), "Output file " + inj_output_filename + " not opened");

                    fout << "ERROR FAIL in kernel execution (" << hipGetErrorString(le) << "); " << std::endl;
                    assert_condition(false,
                                     "ERROR FAIL in kernel execution (" + std::string(hipGetErrorString(le)) + "); ");
                }
                verbose_printf("\n index: ", kernel_id, "; kernel_name: ", kname, "\n");
                kernel_id++; // always increment kernel_id on kernel exit

                // hipDeviceSynchronize();
                pthread_mutex_unlock(&mutex);
            }
        }
    }
}

void nvbit_at_term() {
    fout << "Report_Summary: ;";
    fout << "kernel_index: " << kernel_id << ";";
    fout << "kernel_name: " << last_kernel << ";";
    fout << " LastPCOffset: " << last_pc_offset << ";";
    fout << " LastOpcode: " << current_instruction_opcode << ";";
    fout << " LastInstSASS: " << last_instruction_sass_str << ";";

    fout << inj_info << std::endl;
    hipDeviceSynchronize();
    for (auto i = 0; i < NUM_ISA_INSTRUCTIONS; i++) {
        if (count_activations_inst[i]) {
            verbose_printf(instTypeNames[i], ":", count_activations_inst[i], "\n");
        }
    }
}

#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <stdint.h>
#include <stdio.h>

#include "nvbit_reg_rw.h"
#include "utils/utils.h"
#include "pf_injector.h"
#include "arch.h"
//#include "globals.h"
//#include "cuPrintf.cu"

//__shared__ char *injectionOut ;

extern "C" __device__ __noinline__ void inject_error(uint64_t piinfo, uint64_t pverbose_device, int destGPRNum, int regval, int numDestGPRs, int compute_cap) { 
				inj_info_t* inj_info = (inj_info_t*)piinfo; 
				uint32_t verbose_device = *((uint32_t *)pverbose_device);
				
				//check performed on the Straming Multiprocessor ID
				uint32_t smid;
				asm("mov.u32 %0, %smid;" :"=r"(smid));
				if (smid != inj_info->injSMID) 	return; // This is not the selected SM. No need to proceed.
				uint32_t WID;
				asm("mov.u32 %0, %warpid;" :"=r"(WID));
				uint32_t LID;
				asm("mov.u32 %0, %laneid;" :"=r"(LID));
				int warpID;				
				warpID=int(inj_info->injThreadID/32);
				int laneID=int(inj_info->injThreadID%32);
				
				if(WID!=warpID) return;
				if(LID!=laneID) return;

				assert(numDestGPRs > 0);
				uint32_t injAfterVal = 0; 
				uint32_t injBeforeVal = nvbit_read_reg(destGPRNum); // read the register value
									
				if (DUMMY || destGPRNum != inj_info->injReg ) { 
								injAfterVal = injBeforeVal;
				
				} else {
					if(inj_info->injStuckat == 1){
								injAfterVal = injBeforeVal | (inj_info->injMask); //OR
								nvbit_write_reg(destGPRNum, injAfterVal);
					}
					else {	
								injAfterVal = injBeforeVal & (~inj_info->injMask);//AND			
								nvbit_write_reg(destGPRNum, injAfterVal);
											
					}
				if(verbose_device)printf("register=%d, before=0x%x, after=0x%x, expected_after=0x%x, mask =0x%x, stuck at %d\n", destGPRNum, injBeforeVal, nvbit_read_reg(destGPRNum), injAfterVal,inj_info->injMask,inj_info->injStuckat);
				
				}
				inj_info->errorInjected = true; 
				atomicAdd((unsigned long long*) &inj_info->injNumActivations, 1LL);  
}


#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdio>
#include <cstdlib>

#include <stdint.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <string>
#include <fstream>
#include <vector>
#include <map>
#include <algorithm>
#include <iostream>
#include <sstream>
#include <iterator>
#include <signal.h>
#include <unistd.h>
#include <unordered_set>
#include <cstdlib>

#include "nvbit_tool.h"
#include "nvbit.h"
#include "utils/utils.h"

#include "globals.h"
#include "pf_injector.h"

using namespace std;

int verbose;
__managed__ int verbose_device;

int limit = INT_MAX;

// injection parameters input filename: This file is created the the script
// that launched error injections
std::string injInputFilename = "nvbitfi-injection-info.txt";
pthread_mutex_t mutex;
//__managed__ inj_info_t inj_info; 
__managed__ inj_info_error_t inj_error_info;

__managed__ muliple_ptr_t Injection_masks;

bool read_file=false;

std::string inj_mode;
std::string kname;
std::string SimEndRes;
int num_threads=0;
//inj_info_t inj_info;

void reset_inj_info() {
        inj_error_info.injSMID=0; // 0 - max SMs
        inj_error_info.injScheduler=0; // 0 - 3
        inj_error_info.injWarpMaskH=0;
        inj_error_info.injWarpMaskL=0; //
        inj_error_info.injThreadMask=0; //0-32
        inj_error_info.injMaskSeed=0;
        inj_error_info.injRegID=0; // injection mask
        inj_error_info.injDimension=0;
        inj_error_info.injStuck_at=0;
        inj_error_info.injInstType=0; // instruction type 
        inj_error_info.injRegOriginal=0;
        inj_error_info.injRegReplacement=0;
        inj_error_info.injNumActivations=0;
        inj_error_info.injNumActivAcc=0;
        inj_error_info.injInstrIdx=0;
        inj_error_info.injInstPC=0;
        inj_error_info.injInstOpcode=NOP;
        inj_error_info.blockDimX=0;
        inj_error_info.blockDimY=0;
        inj_error_info.blockDimZ=0;
        inj_error_info.gridDimX=0;
        inj_error_info.gridDimY=0;
        inj_error_info.gridDimZ=0;
        inj_error_info.maxregcount=0;
        inj_error_info.num_threads=0;    
        inj_error_info.TotKerInstr=0;
        inj_error_info.TotAppInstr=0;
        inj_error_info.maxPredReg=-1;    
        inj_error_info.errorInjected = false;
}

void print_inj_info() {
        assert(fout.good());
        //std::cout << "InstType=" << inj_info.injInstType << ", SMID=" << inj_info.injSMID<< ", LaneID=" << inj_info.injLaneID;
        
        //std::cout << ", Mask=" << inj_info.injMask << std::endl;
}

// Parse error injection site info from a file. This should be done on host side.
void parse_paramsIRA(std::string filename) {
        static bool parse_flag = false; // file will be parsed only once - performance enhancement
        if (!parse_flag) {
                parse_flag = true;
                reset_inj_info(); 
                float random=0;
                std::ifstream ifs (filename.c_str(), std::ifstream::in);
                if (ifs.is_open()) {
                        
                        ifs >> inj_error_info.injSMID;
                        ifs >> inj_error_info.injScheduler;
                        ifs >> inj_error_info.injWarpMaskH;
                        ifs >> inj_error_info.injWarpMaskL;
                        ifs >> inj_error_info.injThreadMask;
                        ifs >> inj_error_info.injMaskSeed;
                        ifs >> inj_error_info.injRegID;

                        assert(inj_error_info.injSMID < 1000); 
                        inj_error_info.injNumActivations=0;
                        inj_error_info.injInstrIdx=-1;
                        inj_error_info.injInstOpcode=NOP;
                        inj_error_info.injInstPC=-1;
                        inj_error_info.num_threads=num_threads;

                } else {
                        printf(" File %s does not exist!", filename.c_str());
                        printf(" This file should contain enough information about the fault site to perform a permanent error injection run: ");
                        printf("Documentation to be deifined...\n"); 
                        assert(false);
                }
                ifs.close();

                if (verbose) {
                        print_inj_info();
                }

                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.Warp_thread_active),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));
                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.warp_thread_mask),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));
                //CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.register_tmp_recovery),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp*MAX_KNAME_SIZE)*sizeof(uint32_t)));
                //CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.register_tmp_recovery),(inj_error_info.num_threads*MAX_KNAME_SIZE)*sizeof(uint32_t)));
                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.register_tmp_recovery),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));

                Injection_masks.num_threads=inj_error_info.num_threads;

                srand(inj_error_info.injMaskSeed);
                for(int i=0;i<(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp);++i){
                    random=rand();
                    Injection_masks.warp_thread_mask[i]=*(int *)&random;
                }
                int idx=0;
                int validW=0;
                int validT=0;
                int integer_mask=0;
                for(int i=0;i<inj_error_info.MaxWarpsPerSM;++i){
                    if (i>31){
                        validW=(inj_error_info.injWarpMaskH>>i)&1;
                    }else{
                        validW=(inj_error_info.injWarpMaskL>>i)&1;
                    }
                    for(int j=0;j<inj_error_info.MaxThreadsPerWarp;++j){   
                        validT= (inj_error_info.injThreadMask>>j)&1;                    
                        random=rand();
                        integer_mask=*(int *)&random;
                        Injection_masks.warp_thread_mask[idx]=integer_mask;
                        Injection_masks.Warp_thread_active[idx]=(validW&validT);
                        idx++;
                    }
                }        
        }
}



// Parse error injection site info from a file. This should be done on host side.
void parse_paramsIAT(std::string filename) {
        static bool parse_flag = false; // file will be parsed only once - performance enhancement
        if (!parse_flag) {
                parse_flag = true;
                reset_inj_info(); 
                float random=0;
                std::ifstream ifs (filename.c_str(), std::ifstream::in);
                if (ifs.is_open()) {
                        
                        ifs >> inj_error_info.injSMID;
                        ifs >> inj_error_info.injScheduler;
                        ifs >> inj_error_info.injWarpMaskH;
                        ifs >> inj_error_info.injWarpMaskL;
                        ifs >> inj_error_info.injThreadMask;
                        ifs >> inj_error_info.injMaskSeed;  // 0: inactive thread 1: active thread   
                        ifs >> inj_error_info.injDimension;
                        ifs >> inj_error_info.injStuck_at;                     

                        assert(inj_error_info.injSMID < 1000); 
                        inj_error_info.injNumActivations=0;
                        inj_error_info.injInstrIdx=-1;
                        inj_error_info.injInstOpcode=NOP;
                        inj_error_info.injInstPC=-1;

                } else {
                        printf(" File %s does not exist!", filename.c_str());
                        printf(" This file should contain enough information about the fault site to perform a permanent error injection run: ");
                        printf("Documentation to be deifined...\n"); 
                        assert(false);
                }
                ifs.close();

                if (verbose) {
                        print_inj_info();
                }

                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.Warp_thread_active),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));
                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.warp_thread_mask),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));

                int idx=0;
                int validW=0;
                int validT=0;
                int integer_mask=0;
                for(int i=0;i<inj_error_info.MaxWarpsPerSM;++i){
                    if (i>31){
                        validW=(inj_error_info.injWarpMaskH>>i)&1;
                    }else{
                        validW=(inj_error_info.injWarpMaskL>>i)&1;
                    }
                    for(int j=0;j<inj_error_info.MaxThreadsPerWarp;++j){   
                        validT= (inj_error_info.injThreadMask>>j)&1;
                        //printf("valid %d, %d\n",idx,validW&validT) ;                   
                        Injection_masks.Warp_thread_active[idx]=(validW&validT);
                        Injection_masks.warp_thread_mask[idx]=idx;
                        idx++;
                    }
                }        
        }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////
// Parse error injection site info from a file. This should be done on host side.
void parse_paramsIAC(std::string filename) {
        static bool parse_flag = false; // file will be parsed only once - performance enhancement
        if (!parse_flag) {
                parse_flag = true;
                reset_inj_info(); 
                float random=0;
                std::ifstream ifs (filename.c_str(), std::ifstream::in);
                if (ifs.is_open()) {
                        
                        ifs >> inj_error_info.injSMID;
                        ifs >> inj_error_info.injScheduler;
                        ifs >> inj_error_info.injWarpMaskH;
                        ifs >> inj_error_info.injWarpMaskL;
                        ifs >> inj_error_info.injThreadMask;
                        ifs >> inj_error_info.injMaskSeed;  // 0: inactive thread 1: active thread            
                        ifs >> inj_error_info.injDimension;            
                        ifs >> inj_error_info.injStuck_at;


                        assert(inj_error_info.injSMID < 1000); 
                        inj_error_info.injNumActivations=0;
                        inj_error_info.injInstrIdx=-1;
                        inj_error_info.injInstOpcode=NOP;
                        inj_error_info.injInstPC=-1;

                } else {
                        printf(" File %s does not exist!", filename.c_str());
                        printf(" This file should contain enough information about the fault site to perform a permanent error injection run: ");
                        printf("Documentation to be deifined...\n"); 
                        assert(false);
                }
                ifs.close();

                if (verbose) {
                        print_inj_info();
                }

                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.Warp_thread_active),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));

                int idx=0;
                int validW=0;
                int validT=0;
                int integer_mask=0;
                for(int i=0;i<inj_error_info.MaxWarpsPerSM;++i){
                    if (i>31){
                        validW=(inj_error_info.injWarpMaskH>>i)&1;
                    }else{
                        validW=(inj_error_info.injWarpMaskL>>i)&1;
                    }
                    for(int j=0;j<inj_error_info.MaxThreadsPerWarp;++j){   
                        validT= (inj_error_info.injThreadMask>>j)&1;
                        //printf("valid %d, %d\n",idx,validW&validT) ;                   
                        Injection_masks.Warp_thread_active[idx]=(validW&validT);
                        idx++;
                    }
                }        
        }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////////////////////////////////////////
// Parse error injection site info from a file. This should be done on host side.
void parse_paramsWV(std::string filename) {
        static bool parse_flag = false; // file will be parsed only once - performance enhancement
        if (!parse_flag) {
                parse_flag = true;
                reset_inj_info(); 
                float random=0;
                std::ifstream ifs (filename.c_str(), std::ifstream::in);
                if (ifs.is_open()) {
                        
                        ifs >> inj_error_info.injSMID;
                        ifs >> inj_error_info.injScheduler;
                        ifs >> inj_error_info.injWarpMaskH;
                        ifs >> inj_error_info.injWarpMaskL;
                        ifs >> inj_error_info.injThreadMask;
                        ifs >> inj_error_info.injRegID;
                        ifs >> inj_error_info.injMaskSeed;  // 0: inactive thread 1: active thread                                    
                        ifs >> inj_error_info.injStuck_at;


                        assert(inj_error_info.injSMID < 1000); 
                        inj_error_info.injNumActivations=0;
                        inj_error_info.injInstrIdx=-1;
                        inj_error_info.injInstOpcode=NOP;
                        inj_error_info.injInstPC=-1;

                } else {
                        printf(" File %s does not exist!", filename.c_str());
                        printf(" This file should contain enough information about the fault site to perform a permanent error injection run: ");
                        printf("Documentation to be deifined...\n"); 
                        assert(false);
                }
                ifs.close();

                if (verbose) {
                        print_inj_info();
                }

                CUDA_SAFECALL(hipMallocManaged(&(Injection_masks.Warp_thread_active),(inj_error_info.MaxWarpsPerSM*inj_error_info.MaxThreadsPerWarp)*sizeof(uint32_t)));

                int idx=0;
                int validW=0;
                int validT=0;
                int integer_mask=0;
                for(int i=0;i<inj_error_info.MaxWarpsPerSM;++i){
                    if (i>31){
                        validW=(inj_error_info.injWarpMaskH>>i)&1;
                    }else{
                        validW=(inj_error_info.injWarpMaskL>>i)&1;
                    }
                    for(int j=0;j<inj_error_info.MaxThreadsPerWarp;++j){   
                        validT= (inj_error_info.injThreadMask>>j)&1;
                        //printf("valid %d, %d\n",idx,validW&validT) ;                   
                        Injection_masks.Warp_thread_active[idx]=(validW&validT);
                        idx++;
                    }
                }        
                inj_error_info.maxPredReg=-1;
        }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////

void update_verbose() {
        static bool update_flag = false; // update it only once - performance enhancement
        if (!update_flag) {
            update_flag = true;
            hipDeviceSynchronize();
            verbose_device = verbose;
            hipDeviceSynchronize();
        }
}

int get_maxregs(hipFunction_t func) {
        int maxregs = -1;
        hipFuncGetAttribute(&maxregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, func);
                
        //hipFuncGetAttribute();
        return maxregs;
}


void report_kernel_results(void){
fout <<"Kernel name: "<<kname<<"; kernel Index: "<< kernel_id
        << "; DeviceName: " << inj_error_info.DeviceName
        << "; MaxThreadsPerSM: " << inj_error_info.MaxThreadsPerSM
        << "; MaxWarpsPerSm: " << inj_error_info.MaxWarpsPerSM
        << "; MaxThreadsPerWarp: " << inj_error_info.MaxThreadsPerWarp
        << "; gridDimX: " << inj_error_info.gridDimX
        << "; gridDimY: " << inj_error_info.gridDimY
        << "; gridDimZ: " << inj_error_info.gridDimZ
        << "; blockDimX: " << inj_error_info.blockDimX
        << "; blockDimY: " << inj_error_info.blockDimY
        << "; blockDimZ: " << inj_error_info.blockDimZ
        << "; NumThreads: " << inj_error_info.num_threads;
        if(inj_error_info.errorInjected==true) 
                fout << "; ErrorInjected: True";
        else
                fout << "; ErrorInjected: False"; 
        fout << "; injSmID: " << inj_error_info.injSMID
        << "; injSchID: " << inj_error_info.injScheduler
        << "; injWarpIDH: " << inj_error_info.injWarpMaskH
        << "; injWarpIDL: " << inj_error_info.injWarpMaskL
        << "; injLaneID: " << inj_error_info.injThreadMask;
        if(inj_mode.compare("IRA")==0 or inj_mode.compare("IR")==0){
                fout << "; injRegField: " << inj_error_info.injRegID
                << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; resMaxRegCount: " << inj_error_info.maxregcount
                << "; resRegOrigNum: " << inj_error_info.injRegOriginal
                << "; resRegRepNum: " << inj_error_info.injRegReplacement
                << "; resNumInstr: " << inj_error_info.TotKerInstr;
        }else if(inj_mode.compare("IAT")==0){               
                fout << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; InjDimention: " << inj_error_info.injDimension
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resNumInstr: " << inj_error_info.TotKerInstr;
        }else if(inj_mode.compare("IAW")==0){
                fout << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; InjDimention: " << inj_error_info.injDimension
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resNumInstr: " << inj_error_info.TotKerInstr;
        }else if(inj_mode.compare("IAC")==0){
                fout << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; InjDimention: " << inj_error_info.injDimension
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resNumInstr: " << inj_error_info.TotKerInstr;
        }else if(inj_mode.compare("WV")==0){
                fout << "; injPredReg: " << inj_error_info.injRegID
                << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resMaxPredReg: " << inj_error_info.KernelPredReg
                << "; resNumInstr: " << inj_error_info.TotKerInstr;
        }else {
        }
        fout << "; NumErrInstExeBefStop: " << inj_error_info.injInstrIdx
        << "; LastPCOffset: 0x" << std::hex << inj_error_info.injInstPC  << std::dec
        << "; LastOpcode: " << instTypeNames[inj_error_info.injInstOpcode]
        << "; TotErrAct: " << inj_error_info.injNumActivations << endl; 

}

void report_summary_results(void){
fout << "=================================================================================" << endl;
fout << "Final Report" <<  endl;
fout << "=================================================================================" << endl;
fout << "Report_Summary: " 
        << "; DeviceName: " << inj_error_info.DeviceName
        << "; MaxThreadsPerSM: " << inj_error_info.MaxThreadsPerSM
        << "; MaxWarpsPerSm: " << inj_error_info.MaxWarpsPerSM
        << "; MaxThreadsPerWarp: " << inj_error_info.MaxThreadsPerWarp
        << "; gridDimX: " << inj_error_info.gridDimX
        << "; gridDimY: " << inj_error_info.gridDimY
        << "; gridDimZ: " << inj_error_info.gridDimZ
        << "; blockDimX: " << inj_error_info.blockDimX
        << "; blockDimY: " << inj_error_info.blockDimY
        << "; blockDimZ: " << inj_error_info.blockDimZ;
        if(inj_error_info.errorInjected==true) 
        fout << "; ErrorInjected: True";
        else
        fout << "; ErrorInjected: False"; 

        fout << "; injSmID: " << inj_error_info.injSMID
        << "; injSchID: " << inj_error_info.injScheduler
        << "; injWarpIDH: " << inj_error_info.injWarpMaskH
        << "; injWarpIDL: " << inj_error_info.injWarpMaskL
        << "; injLaneID: " << inj_error_info.injThreadMask;
        if(inj_mode.compare("IRA")==0 or inj_mode.compare("IR")==0){
                fout << "; injRegField: " << inj_error_info.injRegID
                << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; resMaxRegCount: " << inj_error_info.maxregcount
                << "; resRegOrigNum: " << inj_error_info.injRegOriginal
                << "; resRegRepNum: " << inj_error_info.injRegReplacement
                << "; resNumInstr: " << inj_error_info.TotAppInstr;
                if (inj_error_info.maxregcount > inj_error_info.injRegReplacement){
                        fout << "; resRegLoc: InsideLims";
                }else{
                        fout << "; resRegLoc: OutsideLims";
                }
        }else if(inj_mode.compare("IAT")==0){               
                fout << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; InjDimention: " << inj_error_info.injDimension
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resNumInstr: " << inj_error_info.TotAppInstr;
        }else if(inj_mode.compare("IAW")==0){
                fout << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; InjDimention: " << inj_error_info.injDimension
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resNumInstr: " << inj_error_info.TotAppInstr;
        }else if(inj_mode.compare("IAC")==0){
                fout << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; InjDimention: " << inj_error_info.injDimension
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resNumInstr: " << inj_error_info.TotAppInstr;
        }else if(inj_mode.compare("WV")==0){
                fout << "; injPredReg: " << inj_error_info.injRegID
                << "; injMaskSeed: " << inj_error_info.injMaskSeed
                << "; injStuck-at: " << inj_error_info.injStuck_at
                << "; resMaxPredReg: " << inj_error_info.maxPredReg
                << "; resNumInstr: " << inj_error_info.TotAppInstr;
        }else {
        }
        fout << "; NumErrInstExeBefStop: " << inj_error_info.injInstrIdx
        << "; LastPCOffset: 0x" << std::hex << inj_error_info.injInstPC  << std::dec
        << "; LastOpcode: " << instTypeNames[inj_error_info.injInstOpcode]
        << "; TotErrAct: " << inj_error_info.injNumActivAcc+inj_error_info.injNumActivations;
        fout << SimEndRes << endl; 
}


void INThandler(int sig) {
        signal(sig, SIG_IGN); // disable Ctrl-C
        fout << "=================================================================================" << endl;
        fout << "Report for: " << kname << "; kernel Index: "<< kernel_id <<  endl;
        fout << "=================================================================================" << endl;
        fout << ":::NVBit-inject-error; ERROR FAIL Detected Singal SIGKILL;" << endl;
        report_kernel_results();
        SimEndRes="; SimEndRes:::ERROR FAIL Detected Singal SIGKILL::: ";
        report_summary_results();
        fout.flush();
        exit(-1);
}


/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
// DO NOT USE UVM (__managed__) variables in this function
void nvbit_at_init() {
        /* just make sure all managed variables are allocated on GPU */
        setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC","1",1);

        /* we get some environment variables that are going to be use to selectively
         * instrument (within a interval of kernel indexes and instructions). By
         * default we instrument everything. */
        if (getenv("TOOL_VERBOSE")) {
                verbose = atoi(getenv("TOOL_VERBOSE"));
        } else {
                verbose = 0;
        }

        if (getenv("INPUT_INJECTION_INFO")) {
                injInputFilename = getenv("INPUT_INJECTION_INFO");
        }
        if (getenv("OUTPUT_INJECTION_LOG")) {
                injOutputFilename = getenv("OUTPUT_INJECTION_LOG");
        }
        if (getenv("INSTRUMENTATION_LIMIT")) {
                limit = atoi(getenv("INSTRUMENTATION_LIMIT"));
        } 

        if(getenv("nvbitPERfi")){
                inj_mode=getenv("nvbitPERfi");
        }else{
                inj_mode="IRA";
        }

        //GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool (1, 2, 3,..)");

        initInstTypeNameMap();

        signal(SIGINT, INThandler); // install Ctrl-C handler

        open_output_file(injOutputFilename);

        //parse_params(injInputFilename);

        if (verbose) printf("nvbit_at_init:end\n");
        //open_profile_file(injectionOut); 
        //injectionOut = "injection-results.txt";
           //fout3 = fopen("injection-results.txt","a");
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;


void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {

        //parse_params(injInputFilename);  // injParams are updated based on injection seed file
        update_verbose();
        
        
        /* Get related functions of the kernel (device function that can be
         * called by the kernel) */
        std::vector<hipFunction_t> related_functions =
                nvbit_get_related_functions(ctx, func);

        /* add kernel itself to the related function vector */
        related_functions.push_back(func);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, 0) ;
        int archmajor = devProp.major; 
        int archminor = devProp.minor;
        int compute_cap = archmajor*10 + archminor;
        /* iterate on function */
        for (auto f : related_functions) {
                /* "recording" function was instrumented, if set insertion failed
                 * we have already encountered this function */
                if (!already_instrumented.insert(f).second) {
                        continue;
                }

                std::string kname = removeSpaces(nvbit_get_func_name(ctx,f));
                /* Get the vector of instruction composing the loaded CUFunction "func" */
                const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

                int maxregs = get_maxregs(f);
                assert(fout.good());
                //assert(fout3.good());
                int k=0;
                //fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";maxregs: " << maxregs << "(" << maxregs << ")" << std::endl;
                for(auto i: instrs)  {
                        std::string opcode = i->getOpcode(); 
                        std::string instTypeStr = extractInstType(opcode); 
                        int instType = instTypeNameMap[instTypeStr]; 
                        if (verbose) printf("extracted instType: %s, ", instTypeStr.c_str());
                        if (verbose) printf("index of instType: %d\n", instTypeNameMap[instTypeStr]);
                        //if ((uint32_t)instType == inj_info.injInstType || inj_info.injInstType == NUM_ISA_INSTRUCTIONS) {
                        
                        //if ((uint32_t)instType == inj_info.injInstType) {
                                if (verbose) { printf("instruction selected for instrumentation: "); i->print(); }

                                // Tokenize the instruction 
                                std::vector<std::string> tokens;
                                std::string buf; // a buffer string
                                std::stringstream ss(i->getSass()); // Insert the string into a stream
                                while (ss >> buf)
                                        tokens.push_back(buf);

                                int destGPRNum = -1;
                                int numDestGPRs = 0;

                                if (tokens.size() > 1) { // an actual instruction that writes to either a GPR or PR register
                                        if (verbose) printf("num tokens = %ld \n", tokens.size());
                                        int start = 1; // first token is opcode string
                                        if (tokens[0].find('@') != std::string::npos) { // predicated instruction, ignore first token
                                                start = 2; // first token is predicate and 2nd token is opcode
                                        }

                                        // Parse the first operand - this is the first destination
                                        int regnum1 = -1;
                                        int regtype = extractRegNo(tokens[start], regnum1);
                                        if (regtype == 0) { // GPR reg
                                                destGPRNum = regnum1;
                                                numDestGPRs = (getOpGroupNum(instType) == G_FP64) ? 2 : 1;

                                                int szStr = extractSize(opcode); 
                                                if (szStr == 128) {
                                                        numDestGPRs = 4; 
                                                } else if (szStr == 64) {
                                                        numDestGPRs = 2; 
                                                }
                                                
                                                if ((uint32_t)destGPRNum ==inj_error_info.injRegID){
                                                    k++;
                                                    fout <<"Kernel name: "<<kname<<"; kernel Index: "<< kernel_id <<"; Num_Activations: " << k <<";"<< std::endl;

                                                    //printf("instType%d\n",instType);
                                                    nvbit_insert_call(i, "inject_error_IRAv2", IPOINT_AFTER);
                                                    nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                                    nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                                    nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);

                                                    nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                                                    if (destGPRNum != -1) {
                                                        nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                                                } else {
                                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                                }
                                                nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers

                                                nvbit_add_call_arg_const_val32(i, compute_cap); // compute_capability
                                                
                                        }
                                        
                                }
                        }
                }
        }
}


void instrument_function_IRA(hipCtx_t ctx, hipFunction_t func) {

        //parse_params(injInputFilename);  // injParams are updated based on injection seed file
        update_verbose();        
        /* Get related functions of the kernel (device function that can be
        * called by the kernel) */
        std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

        /* add kernel itself to the related function vector */
        related_functions.push_back(func);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, 0) ;
        int archmajor = devProp.major; 
        int archminor = devProp.minor;
        int compute_cap = archmajor*10 + archminor;        
        /* iterate on function */
        for (auto f : related_functions) {                
                /* "recording" function was instrumented, if set insertion failed
                        * we have already encountered this function */
                if (!already_instrumented.insert(f).second) {
                        continue;
                }
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Begins Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;

                std::string kname = removeSpaces(nvbit_get_func_name(ctx,f));
                /* Get the vector of instruction composing the loaded CUFunction "func" */
                const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

                int maxregs = get_maxregs(f);
                inj_error_info.maxregcount=maxregs;
                assert(fout.good());
                //assert(fout3.good());
                int k=0;
                int instridx=0;
                inj_error_info.TotKerInstr=0;
                //fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";maxregs: " << maxregs << "(" << maxregs << ")" << std::endl;
                for(auto i: instrs)  {
                        std::string opcode = i->getOpcode(); 
                        std::string instTypeStr = i->getOpcodeShort();
                        int instType = instTypeNameMap[instTypeStr]; 

                        if (verbose) printf("extracted instType: %s, ", instTypeStr.c_str());
                        if (verbose) printf("index of instType: %d\n", instTypeNameMap[instTypeStr]);
                        //if ((uint32_t)instType == inj_info.injInstType || inj_info.injInstType == NUM_ISA_INSTRUCTIONS) {
                        
                        //if ((uint32_t)instType == inj_info.injInstType) {
                        if (verbose) { printf("instruction selected for instrumentation: "); i->print(); }

                        int destGPRNum = -1;
                        int replGPRNum = -1;
                        int numDestGPRs = 0;
                        fout << "0x" << std::hex << i->getOffset() << ":::" << i->getSass()  << std::dec << std::endl;
                        if (i->getNumOperands() > 1) { // an actual instruction that writes to either a GPR or PR register
                                // Parse the first operand - this is the the destination register field
                                const InstrType::operand_t *dst= i->getOperand(0);
                                if(dst->type == InstrType::OperandType::REG ) { // GPR reg as a destination                                      
                                        numDestGPRs = (getOpGroupNum(instType) == G_FP64) ? 2 : 1;
                                        int szStr = i->getSize()*8; 
                                        if (szStr == 128) {
                                                numDestGPRs = 4; 
                                        } else if (szStr == 64) {
                                                numDestGPRs = 2; 
                                        }                                        
                                        if(inj_error_info.injRegID==0 ){ // and instType!=MOV inject when it is the destination register as target 
                                                destGPRNum = dst->u.reg.num;
                                                inj_error_info.injRegOriginal=destGPRNum;
                                                inj_error_info.injRegReplacement = inj_error_info.injRegOriginal ^ inj_error_info.injMaskSeed;
                                                replGPRNum = inj_error_info.injRegReplacement;
                                                k++;
                                                instridx++; 
                                                inj_error_info.TotKerInstr++;
                                                //fout <<"Kernel name: "<<kname<<"; kernel Index: "<< kernel_id <<"; Num_Activations: " << k <<";"<< std::endl;
                                                //fout << "Instr Intrumented: " << i->getSass();

                                                fout << "0x" << std::hex << i->getOffset() << "; " << i->getSass() << std::dec << " instrumented intruction; ";
                                                fout << "Target_reg_field: "<< inj_error_info.injRegID
                                                        << "; Max_reg_count: "<< inj_error_info.maxregcount
                                                        << "; Original_register: "<< inj_error_info.injRegOriginal
                                                        <<"; Replacement_register: "<< inj_error_info.injRegReplacement
                                                        << "; Error Mask: " << inj_error_info.injMaskSeed
                                                        << "; NumThreads: " << inj_error_info.num_threads << endl;

                                                nvbit_insert_call(i, "inject_error_IRA_dst", IPOINT_AFTER);
                                                nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                                nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                                nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);
                                                nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                                                nvbit_add_call_arg_const_val32(i, replGPRNum); // destination GPR register number
                                                if (destGPRNum != -1) {
                                                        nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                                                } else {
                                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                                }
                                                nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers
                                                nvbit_add_call_arg_const_val32(i, compute_cap); // compute_capability
                                                nvbit_add_call_arg_const_val32(i, instridx);
                                                nvbit_add_call_arg_const_val32(i, i->getOffset());
                                                nvbit_add_call_arg_const_val32(i, instType);
                                                
                                        }else if(inj_error_info.injRegID>0){
                                                int reg_src[5];
                                                int cnt = 0;
                                                        for (int idx = 1; idx < i->getNumOperands(); idx++) {
                                                        const InstrType::operand_t *op = i->getOperand(idx);
                                                        if(op->type == InstrType::OperandType::REG){
                                                        reg_src[cnt]=op->u.reg.num;
                                                        cnt++; 
                                                        }                      
                                                }
                                                if(((uint32_t)cnt)>=inj_error_info.injRegID){
                                                        destGPRNum = (uint32_t)reg_src[inj_error_info.injRegID-1];
                                                        inj_error_info.injRegOriginal=destGPRNum;
                                                        inj_error_info.injRegReplacement = inj_error_info.injRegOriginal ^ inj_error_info.injMaskSeed;
                                                        replGPRNum = inj_error_info.injRegReplacement;
                                                        k++; 
                                                        instridx++;  
                                                        inj_error_info.TotKerInstr++;
                                                        //fout <<"Kernel name: "<<kname<<"; kernel Index: "<< kernel_id <<"; Num_Activations: " << k <<";"<< std::endl;
                                                        //fout << "Instr Intrumented: " << i->getSass();
                                                        fout << "0x" << std::hex << i->getOffset() << "; " << i->getSass() << std::dec << " instrumented intruction; ";
                                                        fout << "Target_reg_field: "<< inj_error_info.injRegID
                                                        << "; Max_reg_count: "<< inj_error_info.maxregcount
                                                        << "; Original_register: "<< inj_error_info.injRegOriginal
                                                        <<"; Replacement_register: "<< inj_error_info.injRegReplacement
                                                        << "; Error Mask: " << inj_error_info.injMaskSeed
                                                        << "; NumThreads: " << inj_error_info.num_threads << endl;
                                                        
                                                        nvbit_insert_call(i, "inject_error_IRA_src_before", IPOINT_BEFORE);
                                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);
                                                        nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                                                        nvbit_add_call_arg_const_val32(i, replGPRNum);
                                                        if (destGPRNum != -1) {
                                                        nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                                                        } else {
                                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                                        }
                                                        nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers
                                                        nvbit_add_call_arg_const_val32(i, compute_cap); // compute_capability
                                                        nvbit_add_call_arg_const_val32(i, instridx); // compute_capability
                                                        nvbit_add_call_arg_const_val32(i, i->getOffset());
                                                        nvbit_add_call_arg_const_val32(i, instType);


                                                        nvbit_insert_call(i, "inject_error_IRA_src_after", IPOINT_AFTER);
                                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);
                                                        nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                                                        nvbit_add_call_arg_const_val32(i, replGPRNum);
                                                        if (destGPRNum != -1) {
                                                        nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                                                        } else {
                                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                                        }
                                                        nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers
                                                        nvbit_add_call_arg_const_val32(i, compute_cap); // compute_capability
                                                        nvbit_add_call_arg_const_val32(i, instridx); // compute_capability
                                                        nvbit_add_call_arg_const_val32(i, i->getOffset());
                                                        nvbit_add_call_arg_const_val32(i, instType);
                                                                                                
                                                }                                
                                        }                                                                        
                                }                            
                        }
                }
                inj_error_info.TotAppInstr+=inj_error_info.TotKerInstr;
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Stops Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;
        }       
}


/* Instrumentation for IAT and IAW error models */
void instrument_function_IAT(hipCtx_t ctx, hipFunction_t func) {

        //parse_params(injInputFilename);  // injParams are updated based on injection seed file
        update_verbose();        
        /* Get related functions of the kernel (device function that can be
        * called by the kernel) */
        std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

        /* add kernel itself to the related function vector */
        related_functions.push_back(func);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, 0) ;
        int archmajor = devProp.major; 
        int archminor = devProp.minor;
        int compute_cap = archmajor*10 + archminor;
        /* iterate on function */
        for (auto f : related_functions) {                
                /* "recording" function was instrumented, if set insertion failed
                        * we have already encountered this function */
                if (!already_instrumented.insert(f).second) {
                        continue;
                }
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Begins Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;

                std::string kname = removeSpaces(nvbit_get_func_name(ctx,f));
                /* Get the vector of instruction composing the loaded CUFunction "func" */
                const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

                int maxregs = get_maxregs(f);
                inj_error_info.maxregcount=maxregs;
                assert(fout.good());
                //assert(fout3.good());
                int k=0;
                int instridx=0;
                int blockDimm=0;
                bool injectInstrunc=false;
                inj_error_info.TotKerInstr=0;
                //fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";maxregs: " << maxregs << "(" << maxregs << ")" << std::endl;
                for(auto i: instrs)  {
                        std::string opcode = i->getOpcode(); 
                        std::string instTypeStr = i->getOpcodeShort();
                        std::string GenOperand;
                        int instType = instTypeNameMap[instTypeStr]; 

                        if (verbose) printf("extracted instType: %s, ", instTypeStr.c_str());
                        if (verbose) printf("index of instType: %d\n", instTypeNameMap[instTypeStr]);
                        //if ((uint32_t)instType == inj_info.injInstType || inj_info.injInstType == NUM_ISA_INSTRUCTIONS) {
                        
                        //if ((uint32_t)instType == inj_info.injInstType) {
                        if (verbose) { printf("instruction selected for instrumentation: "); i->print(); }

                        int destGPRNum = -1;
                        int replGPRNum = -1;
                        int numDestGPRs = 0;
                        injectInstrunc=false;
                        fout << "0x" << std::hex << i->getOffset() << ":::" << i->getSass()  << std::dec << std::endl;
                        if (i->getNumOperands() > 1) { // an actual instruction that writes to either a GPR or PR register
                                // Parse the first operand - this is the the destination register field                                
                                for (int idx=0;idx<i->getNumOperands();++idx){
                                        const InstrType::operand_t *dst= i->getOperand(idx);                                        
                                        if(dst->type == InstrType::OperandType::GENERIC ) { // GPR reg as a destination                                                                                      
                                                GenOperand=dst->str;
                                                size_t found = GenOperand.rfind("TID.X");
                                                if (found != string::npos){
                                                        //blockDimm=inj_error_info.blockDimX-1;
                                                        blockDimm=0;
                                                        injectInstrunc=true;
                                                        //printf("Found: %d; \n",found);
                                                }                                                                                        
                                                found = GenOperand.rfind("TID.Y");
                                                if (found != string::npos){
                                                        //blockDimm=inj_error_info.blockDimY-1;
                                                        blockDimm=1;
                                                        injectInstrunc=true;         
                                                        //printf("Found: %d; \n",found); 
                                                }
                                                                                                
                                                found = GenOperand.rfind("TID.Z");
                                                if (found != string::npos){
                                                        //blockDimm=inj_error_info.blockDimZ-1;
                                                        blockDimm=2;
                                                        injectInstrunc=true;
                                                        //printf("Found: %d; \n",found);
                                                }  
                                                /*                                                                                   
                                                found = GenOperand.rfind("CTAID.X");
                                                if (found != string::npos){
                                                        blockDimm=inj_error_info.gridDimX-1;
                                                        injectInstrunc=true;         
                                                        //printf("Found: %d; \n",found); 
                                                }                                                
                                                found = GenOperand.rfind("CTAID.Y");
                                                if (found != string::npos){
                                                        blockDimm=inj_error_info.gridDimY-1;
                                                        injectInstrunc=true;         
                                                        //printf("Found: %d; \n",found); 
                                                }                                                
                                                found = GenOperand.rfind("CTAID.Z");
                                                if (found != string::npos){
                                                        blockDimm=inj_error_info.blockDimZ-1;
                                                        injectInstrunc=true;
                                                        //printf("Found: %d; \n",found);
                                                }*/
                                                                                                                                                                
                                        }        
                                }                                                                                                 
                                if(injectInstrunc==true && inj_error_info.injDimension==blockDimm){
                                        printf("string: %s; blockDimm: %d\n",GenOperand.c_str(),blockDimm); 
                                        fout << "0x" << std::hex << i->getOffset() << "; " << i->getSass() << std::dec << " instrumented intruction; " << endl;
                                        const InstrType::operand_t *dst= i->getOperand(0);
                                        destGPRNum=dst->u.reg.num;
                                        numDestGPRs=1;
                                        instridx++;  
                                        inj_error_info.TotKerInstr++;
                                        nvbit_insert_call(i, "inject_error_IAT", IPOINT_AFTER);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);
                                        nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                                        if (destGPRNum != -1) {
                                        nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                                        } else {
                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                        }
                                        nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers
                                        nvbit_add_call_arg_const_val32(i, blockDimm); // compute_capability
                                        nvbit_add_call_arg_const_val32(i, instridx); // compute_capability
                                        nvbit_add_call_arg_const_val32(i, i->getOffset());
                                        nvbit_add_call_arg_const_val32(i, instType);
                                        
                                }

                        }

                }
                inj_error_info.TotAppInstr+=inj_error_info.TotKerInstr;
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Stops Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;
        }       
}

/* Instrumentation for IAC*/
void instrument_function_IAC(hipCtx_t ctx, hipFunction_t func) {

        //parse_params(injInputFilename);  // injParams are updated based on injection seed file
        update_verbose();        
        /* Get related functions of the kernel (device function that can be
        * called by the kernel) */
        std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

        /* add kernel itself to the related function vector */
        related_functions.push_back(func);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, 0) ;
        int archmajor = devProp.major; 
        int archminor = devProp.minor;
        int compute_cap = archmajor*10 + archminor;
        /* iterate on function */
        for (auto f : related_functions) {                
                /* "recording" function was instrumented, if set insertion failed
                        * we have already encountered this function */
                if (!already_instrumented.insert(f).second) {
                        continue;
                }
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Begins Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;

                std::string kname = removeSpaces(nvbit_get_func_name(ctx,f));
                /* Get the vector of instruction composing the loaded CUFunction "func" */
                const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

                int maxregs = get_maxregs(f);
                inj_error_info.maxregcount=maxregs;
                assert(fout.good());
                //assert(fout3.good());
                int k=0;
                int instridx=0;
                int gridDimm=0;
                bool injectInstrunc=false;
                inj_error_info.TotKerInstr=0;
                //fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";maxregs: " << maxregs << "(" << maxregs << ")" << std::endl;
                for(auto i: instrs)  {
                        std::string opcode = i->getOpcode(); 
                        std::string instTypeStr = i->getOpcodeShort();
                        std::string GenOperand;
                        int instType = instTypeNameMap[instTypeStr]; 

                        if (verbose) printf("extracted instType: %s, ", instTypeStr.c_str());
                        if (verbose) printf("index of instType: %d\n", instTypeNameMap[instTypeStr]);
                        //if ((uint32_t)instType == inj_info.injInstType || inj_info.injInstType == NUM_ISA_INSTRUCTIONS) {
                        
                        //if ((uint32_t)instType == inj_info.injInstType) {
                        if (verbose) { printf("instruction selected for instrumentation: "); i->print(); }

                        int destGPRNum = -1;
                        int replGPRNum = -1;
                        int numDestGPRs = 0;
                        injectInstrunc=false;
                        fout << "0x" << std::hex << i->getOffset() << ":::" << i->getSass()  << std::dec << std::endl;
                        if (i->getNumOperands() > 1) { // an actual instruction that writes to either a GPR or PR register
                                // Parse the first operand - this is the the destination register field                                
                                for (int idx=0;idx<i->getNumOperands();++idx){
                                        const InstrType::operand_t *dst= i->getOperand(idx);                                        
                                        if(dst->type == InstrType::OperandType::GENERIC ) { // GPR reg as a destination                                                                                      
                                                GenOperand=dst->str;                                                                                 
                                                size_t found = GenOperand.rfind("CTAID.X");
                                                if (found != string::npos){
                                                        gridDimm=0;
                                                        injectInstrunc=true;         
                                                        //printf("Found: %d; \n",found); 
                                                }                                                
                                                found = GenOperand.rfind("CTAID.Y");
                                                if (found != string::npos){
                                                        gridDimm=1;
                                                        injectInstrunc=true;         
                                                        //printf("Found: %d; \n",found); 
                                                }                                                
                                                found = GenOperand.rfind("CTAID.Z");
                                                if (found != string::npos){
                                                        gridDimm=2;
                                                        injectInstrunc=true;
                                                        //printf("Found: %d; \n",found);
                                                }
                                                                                                                                                                
                                        }        
                                }                                                                                                 
                                if(injectInstrunc==true && inj_error_info.injDimension==gridDimm){
                                        printf("string: %s; blockDimm: %d\n",GenOperand.c_str(),gridDimm); 
                                        fout << "0x" << std::hex << i->getOffset() << "; " << i->getSass() << std::dec << " instrumented intruction; " << endl;
                                        const InstrType::operand_t *dst= i->getOperand(0);
                                        destGPRNum=dst->u.reg.num;
                                        numDestGPRs=1;
                                        instridx++; 
                                        inj_error_info.TotKerInstr++;
                                        nvbit_insert_call(i, "inject_error_IAC", IPOINT_AFTER);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);
                                        nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                                        if (destGPRNum != -1) {
                                        nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                                        } else {
                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                        }
                                        nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers
                                        nvbit_add_call_arg_const_val32(i, gridDimm); // compute_capability
                                        nvbit_add_call_arg_const_val32(i, instridx); // compute_capability
                                        nvbit_add_call_arg_const_val32(i, i->getOffset());
                                        nvbit_add_call_arg_const_val32(i, instType);
                                         
                                }

                        }

                }
                inj_error_info.TotAppInstr+=inj_error_info.TotKerInstr;
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Stops Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;
        }       
}


/* Instrumentation for IPP*/
void instrument_function_WV(hipCtx_t ctx, hipFunction_t func) {

        //parse_params(injInputFilename);  // injParams are updated based on injection seed file
        update_verbose();        
        /* Get related functions of the kernel (device function that can be
        * called by the kernel) */
        std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

        /* add kernel itself to the related function vector */
        related_functions.push_back(func);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, 0) ;
        int archmajor = devProp.major; 
        int archminor = devProp.minor;
        int compute_cap = archmajor*10 + archminor;
        /* iterate on function */
        for (auto f : related_functions) {                
                /* "recording" function was instrumented, if set insertion failed
                        * we have already encountered this function */
                if (!already_instrumented.insert(f).second) {
                        continue;
                }
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Begins Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;

                std::string kname = removeSpaces(nvbit_get_func_name(ctx,f));
                /* Get the vector of instruction composing the loaded CUFunction "func" */
                const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

                int maxregs = get_maxregs(f);
                inj_error_info.maxregcount=maxregs;
                assert(fout.good());
                //assert(fout3.good());
                int k=0;
                int instridx=0;
                int gridDimm=0;
                bool injectInstrunc=false;
                inj_error_info.KernelPredReg=-1;
                inj_error_info.TotKerInstr=0;
                //fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";maxregs: " << maxregs << "(" << maxregs << ")" << std::endl;
                for(auto i: instrs)  {
                        std::string opcode = i->getOpcode(); 
                        std::string instTypeStr = i->getOpcodeShort();
                        std::string GenOperand;
                        int instType = instTypeNameMap[instTypeStr]; 

                        if (verbose) printf("extracted instType: %s, ", instTypeStr.c_str());
                        if (verbose) printf("index of instType: %d\n", instTypeNameMap[instTypeStr]);
                        //if ((uint32_t)instType == inj_info.injInstType || inj_info.injInstType == NUM_ISA_INSTRUCTIONS) {
                        
                        //if ((uint32_t)instType == inj_info.injInstType) {
                        if (verbose) { printf("instruction selected for instrumentation: "); i->print(); }

                        int destGPRNum = -1;
                        int replGPRNum = -1;
                        int numDestGPRs = 0;
                        int predicateNum=-1;
                        int tracePredRegs=0;
                        injectInstrunc=false;
                        fout << "0x" << std::hex << i->getOffset() << ":::" << i->getSass()  << std::dec << std::endl;
                        //i->printDecoded();
                        printf("%s %d\n",i->getSass(),i->hasPred()==true ? i->getPredNum():-1);
                        if (i->getNumOperands() > 1) { // an actual instruction that writes to either a GPR or PR register
                                // Parse the first operand - this is the the destination register field                                
                                printf("%s\n",i->getSass());
                                for (int idx=0;idx<i->getNumOperands();++idx){
                                        const InstrType::operand_t *dst= i->getOperand(idx);                                        
                                        if(dst->type == InstrType::OperandType::PRED && injectInstrunc==false && idx==0) { // GPR reg as a destination                                                                                      
                                                GenOperand=dst->str;  
                                                predicateNum=dst->u.pred.num;
                                                //predicateNum=6;                                                                                                                               
                                                printf("pred:%s; num: %d\n", dst->str, dst->u.pred.num); 
                                                injectInstrunc=true;                                                                                                                                                                                                               
                                        }  
                                        if(dst->type == InstrType::OperandType::PRED && idx==0){
                                                tracePredRegs=dst->u.pred.num;
                                                if(inj_error_info.KernelPredReg<(tracePredRegs)){
                                                        inj_error_info.KernelPredReg=(tracePredRegs);
                                                }
                                        }                                              
                                }                                                                                                 
                                if(injectInstrunc==true and inj_error_info.injRegID==predicateNum){
                                        printf("string: %s; blockDimm: %d\n",GenOperand.c_str(),gridDimm); 
                                        fout << "0x" << std::hex << i->getOffset() << "; " << i->getSass() << std::dec << " instrumented intruction; " << endl;
                                        const InstrType::operand_t *dst= i->getOperand(0);
                                        destGPRNum=dst->u.reg.num;
                                        numDestGPRs=1;
                                        instridx++; 
                                        inj_error_info.TotKerInstr++; 
                                        nvbit_insert_call(i, "inject_error_WV", IPOINT_AFTER);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&inj_error_info);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&Injection_masks);
                                        nvbit_add_call_arg_const_val64(i, (uint64_t)&verbose_device);
                                        nvbit_add_call_arg_const_val32(i, predicateNum); // destination GPR register number                                        
                                        if (predicateNum != -1) {
                                        nvbit_add_call_arg_pred_val_at(i, 1); // destination GPR register val
                                        } else {
                                        nvbit_add_call_arg_const_val32(i, (unsigned int)-1); // destination GPR register val 
                                        }
                                        nvbit_add_call_arg_pred_reg(i);
                                        nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers
                                        nvbit_add_call_arg_const_val32(i, gridDimm); // compute_capability
                                        nvbit_add_call_arg_const_val32(i, instridx); // compute_capability
                                        nvbit_add_call_arg_const_val32(i, i->getOffset());
                                        nvbit_add_call_arg_const_val32(i, instType);                                        
                                }

                        }

                }
                
                if(inj_error_info.maxPredReg<inj_error_info.KernelPredReg){
                        inj_error_info.maxPredReg=inj_error_info.KernelPredReg;
                }
                inj_error_info.TotAppInstr+=inj_error_info.TotKerInstr;
                fout << "=================================================================================" << endl;
                fout << "The Instrumentation step Stops Here: " << removeSpaces(nvbit_get_func_name(ctx,f)) << endl;
                fout << "=================================================================================" << endl;
        }       
        
}

/* This call-back is triggered every time a CUDA event is encountered.
 * Here, we identify CUDA kernel launch events and reset the "counter" before
 * th kernel is launched, and print the counter after the kernel has completed
 * (we make sure it has completed by using hipDeviceSynchronize()). To
 * selectively run either the original or instrumented kernel we used
 * nvbit_enable_instrumented() before launching the kernel. */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                const char *name, void *params, hipError_t *pStatus) {
        /* Identify all the possible CUDA launch events */
        if (cbid == API_CUDA_cuLaunch ||
                        cbid == API_CUDA_cuLaunchKernel_ptsz ||
                        cbid == API_CUDA_cuLaunchGrid ||
                        cbid == API_CUDA_cuLaunchGridAsync || 
                        cbid == API_CUDA_cuLaunchKernel) {

                /* cast params to cuLaunch_params since if we are here we know these are
                 * the right parameters type */
                //cuLaunch_params * p = (cuLaunch_params *) params;    
                auto *p = (cuLaunch_params *) params;
                auto *p1 = (cuLaunchKernel_params *) params;             
                num_threads  = p1->gridDimX * p1->gridDimY * p1->gridDimZ * p1->blockDimX * p1->blockDimY * p1->blockDimZ;                              
                if(!is_exit) {
                    if(read_file==false){
                        int MaxThreadsPerSM=0;
                        hipDevice_t device;
                        hipDeviceGet(&device, 0);
                        hipDeviceGetAttribute(&MaxThreadsPerSM,hipDeviceAttributeMaxThreadsPerMultiProcessor,device);
                        hipDeviceGetName(inj_error_info.DeviceName,256,device);
                        inj_error_info.MaxThreadsPerWarp=32;
                        inj_error_info.MaxThreadsPerSM=MaxThreadsPerSM;
                        inj_error_info.MaxWarpsPerSM=MaxThreadsPerSM/inj_error_info.MaxThreadsPerWarp;
                        if(inj_mode.compare("IRA")==0 || inj_mode.compare("IR")==0){
                                parse_paramsIRA(injInputFilename);                                
                        }else if(inj_mode.compare("IAT")==0 || inj_mode.compare("IAW")==0){
                                parse_paramsIAT(injInputFilename); 
                        }else if(inj_mode.compare("IAC")==0){
                                parse_paramsIAC(injInputFilename); 
                        }else if(inj_mode.compare("WV")==0){
                                parse_paramsWV(injInputFilename); 
                        }else{
                                assert(1==0);
                        }         
                                       
                        if (verbose) printf("read file list done..\n");               
                        read_file=true;
                    } 
                        inj_error_info.gridDimX=p1->gridDimX;
                        inj_error_info.gridDimY=p1->gridDimY;
                        inj_error_info.gridDimZ=p1->gridDimZ;
                        inj_error_info.blockDimX=p1->blockDimX;
                        inj_error_info.blockDimY=p1->blockDimY;
                        inj_error_info.blockDimZ=p1->blockDimZ;
                        inj_error_info.num_threads  = num_threads;

                        pthread_mutex_lock(&mutex);
                        if (kernel_id < limit) {
                            hipDeviceSynchronize();
                            fflush (stdout);
                            fclose (stdout);
                            freopen ("nvbit_stdout.txt", "a", stdout);
                                kname = removeSpaces(nvbit_get_func_name(ctx,p->f));
                                if(inj_mode.compare("IRA")==0){
                                        instrument_function_IRA(ctx, p->f);
                                }else if(inj_mode.compare("IAT")==0 || inj_mode.compare("IAW")==0){
                                        instrument_function_IAT(ctx, p->f);
                                }else if(inj_mode.compare("IAC")==0) {
                                        instrument_function_IAC(ctx, p->f);
                                }else if(inj_mode.compare("WV")==0) {
                                        instrument_function_WV(ctx, p->f);
                                }else{
                                        assert(1==0);
                                }
                            hipDeviceSynchronize();
                            fout << "=================================================================================" << endl;
                            fout << "Running instrumented Kernel: " << removeSpaces(nvbit_get_func_name(ctx,p->f)) << "; kernel Index: "<< kernel_id << endl;
                            fout << "..............." << endl;
                            fout << "=================================================================================" << endl;
                            nvbit_enable_instrumented(ctx, p->f, true); // run the instrumented version
                            hipDeviceSynchronize();                            
                        } else {
                            nvbit_enable_instrumented(ctx, p->f, false); // do not use the instrumented version
                        }

                }  else {
                        if (kernel_id < limit) {
                                if (verbose) printf("is_exit\n"); 
                                hipDeviceSynchronize();
                                fflush (stdout);
                                fclose (stdout);
                                freopen ("stdout.txt", "a", stdout);

                                hipError_t le = hipGetLastError();
                                kname = removeSpaces(nvbit_get_func_name(ctx,p->f));
                                //int num_ctas = 0;
                                //int num_threads = 0;//added
                                if ( cbid == API_CUDA_cuLaunchKernel_ptsz ||
                                                cbid == API_CUDA_cuLaunchKernel) {
                                        //cuLaunchKernel_params * p2 = (cuLaunchKernel_params*) params;
                                        //num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
                                        //num_threads = num_ctas * p2->blockDimX * p2->blockDimY * p2->blockDimZ; //added 

                                }
                                assert(fout.good());                            
                                fout << "=================================================================================" << endl;
                                fout << "Report for: " << kname << "; kernel Index: "<< kernel_id <<  endl;
                                fout << "=================================================================================" << endl;
                                if ( hipSuccess != le ) {
                                        assert(fout.good());                                        
                                        std::string cuerr = hipGetErrorString(le);
                                        fout << "ERROR FAIL in kernel execution (" << cuerr << "); " <<std::endl;
                                        report_kernel_results();                                       
                                        SimEndRes = "; SimEndRes:::ERROR FAIL in kernel execution (" + cuerr + "):::";                                        
                                        exit(1); // let's exit early 
                                }
                                //fout << "inspecting: "<< kname <<"; thread : "<<  inj_info.injThreadID <<"; Register : "<< inj_info.injReg<<";  Mask : "<<inj_info.injMask<<"; SMID : "<<inj_info.injSMID<< "; Stuck at : "<<inj_info.injStuckat  << "; index: " << kernel_id << ";" <<std::endl;
                                report_kernel_results();
                                SimEndRes = "; SimEndRes:::PASS without fails:::";
                                inj_error_info.injNumActivAcc+= inj_error_info.injNumActivations;
                                inj_error_info.injNumActivations=0;

                                if (verbose) printf("\n index: %d; kernel_name: %s; \n", kernel_id, kname.c_str());
                                kernel_id++; // always increment kernel_id on kernel exit

                                //hipDeviceSynchronize();
                                pthread_mutex_unlock(&mutex);                            
                            
                        }
                }
        }
}

void nvbit_at_term() { 
    if (verbose) printf("nvbit_at_term:start\n");
    assert(fout.good());
    report_summary_results();
    if (verbose) printf("nvbit_at_term:end\n");
} 

